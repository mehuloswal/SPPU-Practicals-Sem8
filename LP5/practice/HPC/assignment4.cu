#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <functional>
// #include <stdlib.h> /* srand, rand */
// #include <time.h>   /* time */

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] + b[id];
}

int main()
{
    int n = 10;

    double *h_a;
    double *h_b;
    double *h_c;

    double *d_a;
    double *d_b;
    double *d_c;

    size_t bytes = n * sizeof(double);

    h_a = (double *)malloc(bytes);
    h_b = (double *)malloc(bytes);
    h_c = (double *)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    for (int i = 0; i < n; i++)
    {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blocksize, gridsize;

    blocksize = 1024;
    gridsize = (int)ceil((float)n / blocksize);

    vecAdd<<gridsize,blocksize>>(d_a,d_b,d_c,n);

    hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);
    

}